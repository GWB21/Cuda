﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_DATA 128

//디바이스 코드... gpu에서 각각의 스레드들이 실행할 명령
__device__ void deviceVecAdd(int* a, int* b, int* c, int tID) {
    if (tID < NUM_DATA) {
        c[tID] = a[tID] + b[tID];
        /*printf("%d + %d = %d from thread %d\n", a[tID], b[tID], c[tID], tID);*/
    }
}

// GPU에서 실행되는 커널 함수
__global__ void vecAdd(int* a, int* b, int* c) {
    int tID = threadIdx.x;  // 스레드 ID를 글로벌에서 생성해서 디바이스코드로 전달하기 위해 threadIdx.x를 사용
    deviceVecAdd(a, b, c, tID);
}

int main(void) {
    int* a, * b, * c;         // 호스트 메모리 포인터
    int* d_a, * d_b, * d_c;   // 디바이스 메모리 포인터
    int memSize = sizeof(int) * NUM_DATA; //메모리사이즈 할당 위해 계산... int => 4byte => 32bit => -2^31 ~ -2^31 + 1 범위가짐

    printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

    // 호스트 메모리 할당 및 초기화
    a = new int[NUM_DATA];
    b = new int[NUM_DATA];
    c = new int[NUM_DATA];
    for (int i = 0; i < NUM_DATA; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    // 디바이스 메모리 할당
    hipMalloc(&d_a, memSize);
    hipMalloc(&d_b, memSize);
    hipMalloc(&d_c, memSize);

    // 호스트 메모리 -> 디바이스 메모리 복사
    // cudaMemcpy(목적지, 출발지, 메모리사이즈,..,복사유형..HostToDevice같은..)
    hipMemcpy(d_a, a, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, memSize, hipMemcpyHostToDevice);

    // GPU 커널 호출
    vecAdd <<<1, NUM_DATA >>> (d_a, d_b, d_c);
    hipDeviceSynchronize();  // GPU 연산 완료 대기

    // 디바이스 메모리 -> 호스트 메모리 복사
    hipMemcpy(c, d_c, memSize, hipMemcpyDeviceToHost);

    // 결과 출력
    /*printf("Vector Addition Result\n");
    for (int i = 0; i < NUM_DATA; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }*/

    // 디바이스 메모리 해제
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // 호스트 메모리 해제
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}